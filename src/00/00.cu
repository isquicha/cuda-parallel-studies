/**
    @brief Compare vector sum calculation functions in CPU vs GPU.
    @file 00.cu
    @author isquicha
    @version 0.1.0
*/

#include <stdio.h>
#include <time.h>

// Cuda headers are on CUDA Toolkit instalation path/VERSION/include
#include "hip/hip_runtime.h"



#define N_THREADS 1024
#define LOOP_TIMES 10000000
/*
    The functions run too fast to see the difference, so to compare
    we run then in a loop.
    In my environment
*/

/**
 * Sum numbers of two vectors on GPU.
 *
 * The result is stored on a third vector.
 *
 * @param A First input vector pointer.
 * @param B Second input vector pointer.
 * @param C Output vector pointer.
 * @return void
 */
__global__ void dMatAdd(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    for (long j = 0; j < LOOP_TIMES; j++)
        C[i] = A[i] + B[i];
}

/**
 * Sum numbers of two vectors on CPU.
 *
 * The result is stored on a third vector.
 *
 * @param A First input vector pointer.
 * @param B Second input vector pointer.
 * @param C Output vector pointer.
 * @return void
 */
void hMatAdd(float *A, float *B, float *C)
{
    for (int i = 0; i < N_THREADS; i++)
    {
        for (long j = 0; j < LOOP_TIMES; j++)
        {
            C[i] = A[i] + B[i];
        }
    }
}

/**
 * Compare two float vectors
 *
 * @param A First input vector pointer.
 * @param B Second input vector pointer.
 * @return true if vectors are equal, false otherwise
 */
bool compare(float *A, float *B)
{
    for (int i = 0; i < N_THREADS; i++)
    {
        if (A[i] != B[i])
        {
            printf(
                "Elements are not equal. Index %d\t\tA: %f\t\tB:%f\n",
                i, A[i], B[i]);
            return false;
        }
    }
    return true;
}

int main(int argc, char const *argv[])
{
    // Timing variables
    clock_t h_begin, h_end, d_begin, d_end;
    double h_time, d_time;

    // Size auxiliar
    size_t size = N_THREADS * sizeof(float);

    // Host memory allocation
    float *hA = (float *)malloc(size);
    float *hB = (float *)malloc(size);
    float *hC = (float *)malloc(size);
    float *hC2 = (float *)malloc(size);
    if (hA == NULL || hB == NULL || hC == NULL || hC2 == NULL)
    {
        printf("Malloc error!\n");
        exit(1);
    }

    // Device memory allocation
    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dB, size);
    hipMalloc((void **)&dC, size);

    // Vectors initialization with some values
    for (int i = 0, j = N_THREADS; i < N_THREADS; i++, j--)
    {
        hA[i] = float(i);
        //printf("hA[%d] = %f\n", i, hA[i]);
        hB[i] = float(j);
    }

    // Host function
    printf("Running Host function\n");
    h_begin = clock();
    hMatAdd(hA, hB, hC);
    h_end = clock();

    // Device function
    printf("Running Device function\n");
    d_begin = clock();
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
    dMatAdd<<<1, N_THREADS>>>(dA, dB, dC);
    hipMemcpy(hC2, dC, size, hipMemcpyDeviceToHost);
    d_end = clock();

    // Results
    h_time = (double)(h_end - h_begin) / CLOCKS_PER_SEC;
    d_time = (double)(d_end - d_begin) / CLOCKS_PER_SEC;

    printf("Running Compare function\n");
    printf("Vectors are equal?: %s\n", compare(hC, hC2) ? "true" : "false");
    printf("CPU: %f seconds\n", h_time);
    printf("GPU: %f seconds\n", d_time);

    // Memory free
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);
    free(hC);
    free(hC2);

    return 0;
}
